#include "hip/hip_runtime.h"
#include "addmul_kernel.h"

#define BLOCK_SIZE 16
#define NUM_THREADS 1024
#define EXPERIMENTAL


template <typename scalar_t>
__global__ void matmul_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> A,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> B,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> result,
    const int m, const int p
)
{
#ifdef EXPERIMENTAL
    // use shared memory technique
    __shared__ scalar_t As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ scalar_t Bs[BLOCK_SIZE][BLOCK_SIZE];

    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;

    scalar_t sum = 0;
    for (int i = 0; i < A.size(1); i += BLOCK_SIZE) {
        if (i + threadIdx.y < A.size(1) && row < m) {
            As[threadIdx.x][threadIdx.y] = A[row][i + threadIdx.y];
        }
        else {
            As[threadIdx.x][threadIdx.y] = 0;
        }
        if (i + threadIdx.x < B.size(0) && col < p) {
            Bs[threadIdx.x][threadIdx.y] = B[i + threadIdx.x][col];
        }
        else {
            Bs[threadIdx.x][threadIdx.y] = 0;
        }
        __syncthreads();

        for (int j = 0; j < BLOCK_SIZE; j++) {
            sum += As[threadIdx.x][j] * Bs[j][threadIdx.y];
        }
        __syncthreads();
    }
    if (row < m && col < p) {
        result[row][col] = sum;
    }
#else
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row >= m || col >= p) return;

    scalar_t sum = 0;
    for (int i = 0; i < A.size(1); i++) {
        sum += A[row][i] * B[i][col];
    }
    result[row][col] = sum;
#endif
}

torch::Tensor matmul_cuda(torch::Tensor A, torch::Tensor B) {

    const int m = A.size(0);
    const int n = A.size(1);
    const int p = B.size(1);
    
    // Create output tensor
    auto result = torch::empty({m, p}, A.options());

    const dim3 blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 gridSize = dim3(DIV_CEIL(m, BLOCK_SIZE), DIV_CEIL(p, BLOCK_SIZE));
  
    // Call the cuda kernel launcher
    AT_DISPATCH_FLOATING_TYPES(A.type(), "matmul_cuda", 
    ([&] {
        matmul_fw_kernel<scalar_t><<<gridSize, blockSize>>>(
            A.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            B.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            result.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            m, p
        );
    }));

    return result;
}


template <typename scalar_t>
__global__ void transpose_kernel(
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> A,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> res,
    const int m, const int p
)
{
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;
#ifdef EXPERIMENTAL
    // use shared memory
    __shared__ scalar_t As[BLOCK_SIZE][BLOCK_SIZE];
    if (row < m && col < p) {
        As[threadIdx.x][threadIdx.y] = A[row][col];
    }
    __syncthreads();

    const int row2 = blockIdx.y * blockDim.y + threadIdx.x;
    const int col2 = blockIdx.x * blockDim.x + threadIdx.y;
    if (row2 < p && col2 < m) {
        res[row2][col2] = As[threadIdx.y][threadIdx.x];
    }

#else
    
    if (row >= m || col >= p) return;

    res[col][row] = A[row][col];

#endif
}


torch::Tensor transpose_cuda(const torch::Tensor A) {

    const int m = A.size(0);
    const int n = A.size(1);
    
    // Create output tensor
    auto result = torch::empty({n, m}, A.options());

    const dim3 blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 gridSize = dim3(DIV_CEIL(m, BLOCK_SIZE), DIV_CEIL(n, BLOCK_SIZE));
  
    // Call the cuda kernel launcher
    AT_DISPATCH_FLOATING_TYPES(A.type(), "matmul_cuda", 
    ([&] {
        transpose_kernel<scalar_t><<<gridSize, blockSize>>>(
            A.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            result.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            m, n
        );
    }));

    return result;
}


template <typename scalar_t>
__global__ void add_inplace_nxp_1xp_kernel(
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> A,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> B,
    const int n, const int p
)
{
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= n || col >= p) return;

    // move B into the shared memory
    __shared__ scalar_t Bs[BLOCK_SIZE];

    /* Bug: the following commented code has wrong dimension calculation */
    // if (threadIdx.y == 0) {
        // Bs[threadIdx.x] = B[col];
    // }
    // __syncthreads();
    // A[row][col] += Bs[threadIdx.x];

    if (threadIdx.x == 0) {
        Bs[threadIdx.y] = B[col];
    }
    __syncthreads();

    A[row][col] += Bs[threadIdx.y];

}


/*
 * A: n x p matrix, B: p-dim vector (1 x p)
 * Use broadcasting. Result store in A.
 */
torch::Tensor add_inplace_nxp_p_cuda(
    const torch::Tensor A, 
    const torch::Tensor B
) {

    const int n = A.size(0);
    const int p = A.size(1);
    
    const dim3 blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 gridSize = dim3(DIV_CEIL(n, BLOCK_SIZE), DIV_CEIL(p, BLOCK_SIZE));
  
    // Call the cuda kernel launcher
    AT_DISPATCH_FLOATING_TYPES(A.type(), "add_inplace_nxp_1xp_cuda", 
    ([&] {
        add_inplace_nxp_1xp_kernel<scalar_t><<<gridSize, blockSize>>>(
            A.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            B.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            n, p
        );
    }));

    return A;
}


template <typename scalar_t>
__global__ void sum_axis_kernel(
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> A,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> res,
    const int axis, const int n, const int m
)
{
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;

    scalar_t sum = 0;
    __shared__ scalar_t sums[NUM_THREADS];

    // along row
    if (axis == 0) {
        for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < n; idx += blockDim.x * gridDim.x) {
            sum += A[idx][col];
        }
        sums[threadIdx.x] = sum;
        __syncthreads();

        // reduce
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                sums[threadIdx.x] += sums[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // write result
        if (threadIdx.x == 0) {
            res[col] = sums[0];
        }
        // if (blockIdx.x * blockDim.x < n) {
        //     if (threadIdx.x == 0) {
        //         atomicAdd(&res[col], sums[0]);
        //     }
        // }
    } else {
        // along col
        for (int idx = threadIdx.y + blockIdx.y * blockDim.y; idx < m; idx += blockDim.y * gridDim.y) {
            sum += A[row][idx];
        }
        sums[threadIdx.y] = sum;
        __syncthreads();

        // reduce
        for (int stride = blockDim.y / 2; stride > 0; stride >>= 1) {
            if (threadIdx.y < stride) {
                sums[threadIdx.y] += sums[threadIdx.y + stride];
            }
            __syncthreads();
        }
        
        // write result
        // if (blockIdx.y * blockDim.y < m) {
        //     if (threadIdx.y == 0) {
        //         atomicAdd(&res[row], sums[0]);
        //     }
        // }
        if (threadIdx.y == 0) {
            res[row] = sums[0];
        }
    }
}


torch::Tensor sum_axis_cuda(const torch::Tensor A, int axis)
{
    const int n = A.size(0);
    const int m = A.size(1);
    
    // Create output tensor
    auto result = torch::zeros({axis == 0 ? m : n}, A.options());

    // create block, grid config
    dim3 blockSize, gridSize;
    if (axis == 0) {
        blockSize = dim3(NUM_THREADS, 1);
        // gridSize = dim3(SQRT_CEIL(n/NUM_THREADS), m);
        gridSize = dim3(1, m);
    }
    else {
        blockSize = dim3(1, NUM_THREADS);
        gridSize = dim3(n, 1);
    }
  
    // Call the cuda kernel launcher
    AT_DISPATCH_FLOATING_TYPES(A.type(), "sum_axis_cuda", 
    ([&] {
        sum_axis_kernel<scalar_t><<<gridSize, blockSize>>>(
            A.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            result.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            axis, n, m
        );
    }));

    return result;
}